#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include<hip/hip_runtime.h>
 
// CUDA kernel. One thread execute 1 workload of C = A + B
__global__ void vecAdd(float *a, float *b, float *c, int n){
    int i = blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x; 
    
    if (i < n)
        c[i] = a[i] * a[i] + b[i] * b[i];
}

#define ARR_SIZE 1024*1024*64 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = ARR_SIZE;
 
    // Host vectors
    float *h_a;
    float *h_b;
    float *h_c;
 
    // Device input vectors
    float *d_a;
    float *d_b;
    float *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
 
    // Allocate memory  on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
 
    // Allocate memory  on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sinf(i);
        h_b[i] = cosf(i);
    }
 
    // Copy from host to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    uint3 blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize.x = 32;
    blockSize.y = 32;
    blockSize.z = 1;

    // total blocks in grid
    gridSize.x = (int)ceil((float)n/(blockSize.x*blockSize.y*blockSize.z));
    gridSize.y = 1;
    gridSize.z = 1;

    //Profile GPU time
    float time_elapsed=0;
    hipEvent_t start,stop;

    hipEventCreate(&start);    
    hipEventCreate(&stop);
    
    // Execute the kernel
    hipEventRecord( start,0);
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipEventRecord( stop,0);
    
    hipEventSynchronize(start); 
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&time_elapsed,start,stop);    
    printf("Costs  %f(ms)\n",time_elapsed);

    // Copy result back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Result must be close to 1
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/(double)n);
 
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // free host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}

 

