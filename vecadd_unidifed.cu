#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include<hip/hip_runtime.h>
 
// CUDA kernel. One thread execute 1 workload of C = A + B
__global__ void vecAdd(float *a, float *b, float *c, int n){
    int i = blockIdx.x*blockDim.x+threadIdx.x; 
    
    if (i < n)
        c[i] = a[i] * a[i] + b[i] * b[i];
}

#define ARR_SIZE (1024*16) 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = ARR_SIZE;
 
    // // Host vectors
    // float *h_a;
    // float *h_b;
    // float *h_c;
 
    // Device input vectors
    float *d_a;
    float *d_b;
    float *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
    // // Allocate memory  on host
    // h_a = (float*)malloc(bytes);
    // h_b = (float*)malloc(bytes);
    // h_c = (float*)malloc(bytes);
 
    // Allocate memory  on GPU
    hipMallocManaged(&d_a, bytes);
    hipMallocManaged(&d_b, bytes);
    hipMallocManaged(&d_c, bytes);
 
    int i;
    // Initialize on host
    for( i = 0; i < n; i++ ) {
        d_a[i] = sinf(i);
        d_b[i] = cosf(i);
    }
 
    // // Copy from host to device
    // cudaMemcpy( d_a, h_a, bytes, cudaMemcpyHostToDevice);
    // cudaMemcpy( d_b, h_b, bytes, cudaMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // total blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();
 
    // Copy result back to host
    //cudaMemcpy( h_c, d_c, bytes, cudaMemcpyDeviceToHost );
 
    // Result must be close to 1
    double sum = 0;
    for(i=0; i<n; i++)
        sum += d_c[i];
    printf("final result: %f\n", sum/(double)n);
 
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // // free host memory
    // free(h_a);
    // free(h_b);
    // free(h_c);
 
    return 0;
}

 
